#include "hip/hip_runtime.h"
__global__ void
Gaussian_CUDA(int* A, int wA, int row, int* temp)
{
        int bx, tx, by, ty;
        int i;
        float tmp;

        // Block index
        bx = blockIdx.x;
        by = blockIdx.y;

        // Thread index
        tx = threadIdx.x;
        ty = threadIdx.y;

        int index = wA * BLOCK_SIZE * by + BLOCK_SIZE * bx + wA * ty + tx;

        int index_col = BLOCK_SIZE * bx + tx;
        int index_row = BLOCK_SIZE * by + ty;

        if (by == 0 && ty == 0)
        temp[index] = A[index + row * wA];
        __syncthreads();

        for(i = row + 1; i < wA; i++)
       {
        if ( by == 0 && ty == 0)
        tmp = A[i * wA + row]/temp[row];
        __syncthreads();


        if ( by == 0 && ty == 0)
        A[i * wA + index] = A[i * wA + index] - temp[index] * tmp;

        __syncthreads();
       }
}
