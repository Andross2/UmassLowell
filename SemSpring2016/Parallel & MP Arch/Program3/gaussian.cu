// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "gaussian_kernel.cu"

#define OUTPUT

void runTest(int argc, char** argv);

double gettime() {
        struct timeval t;
        gettimeofday(&t,NULL);
        return t.tv_sec+t.tv_usec*1e-6;
}

int main(int argc, char** argv)
{
        runTest(argc, argv);
}

void runTest(int argc, char** argv)
{
        int dim;

        if (argc == 2)
        {
                dim = atoi(argv[1]);
        }
        else{
                printf("Wrong Usage\n");
                exit(1);
        }
// allocate host memory for matrices A and B
        unsigned int size_A = dim * dim;
        unsigned int mem_size_A = sizeof(int) * size_A;
        int* h_A = (int*) malloc(mem_size_A);


        // initialize host memory, generate a test case such as below
        //   1 1 1 1 ..
        //   1 2 2 2 ..
        //   1 2 3 3 ..
        //   1 2 3 4 ..
        //   ..........

        for( int i = 0; i < dim; i++){
                for (int j = 0 ; j < dim - i; j++){
                        h_A[j + i + i * dim] = i + 1;
                        h_A[j * dim + i + i * dim] = i + 1;
                }
        }

        //display the test case
        /*
        for ( int m = 0 ; m < dim; m++){
                for ( int n = 0 ; n < dim; n++){
                        printf("%d ", h_A[m * dim + n]);
                }
                printf("\n");
        }
        */


        // allocate device memory for the matrix A
        int* d_A;
        hipMalloc((void **) &d_A, mem_size_A);

        int* temp; //temporary array to store dim number of integer elements
        hipMalloc((void **) &temp,(dim * sizeof(int)));

        // copy host memory to device
        double timer1 = gettime();
        hipMemcpy(d_A,h_A,mem_size_A,hipMemcpyHostToDevice);


        // setup execution parameters
        dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid(dim / threads.x, dim / threads.y);

        // execute the kernel
        for ( int i = 0 ; i < dim ; i++){
                Gaussian_CUDA<<< grid, threads >>>(d_A, dim, i, temp);
        }

        // copy result from device to host
        hipMemcpy((void *)h_A,(void *)d_A,mem_size_A,hipMemcpyDeviceToHost);

        double timer2 = gettime();
        printf("GPU time = %lf\n",(timer2-timer1)*1000);

#ifdef OUTPUT

        //the result should be I(dim*dim)
        for ( int m = 0 ; m < dim; m++){
                for ( int n = 0 ; n < dim; n++){
                        printf("%d ", h_A[m * dim + n]);
                }
                printf("\n");
        }
#endif

        free(h_A);
        hipFree(d_A);
        hipFree(temp);
}

